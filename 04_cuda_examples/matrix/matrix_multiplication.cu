#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>

#define BLOCK_SIZE 16

// Simplified GPU kernel for matrix multiplication
__global__ void matrixMulKernel(float *A, float *B, float *C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;

    if (row < M && col < K) {
        // Compute the dot product for the element C[row, col]
        for (int k = 0; k < N; k++) {
            sum += A[row * N + k] * B[k * K + col];
        }
        C[row * K + col] = sum;
    }
}

// CPU implementation
void matrixMulCPU(float *A, float *B, float *C, int M, int N, int K) {
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < K; j++) {
            float sum = 0.0f;
            for (int k = 0; k < N; k++) {
                sum += A[i * N + k] * B[k * K + j];
            }
            C[i * K + j] = sum;
        }
    }
}

int main() {
    const int M = 1024;  // A rows
    const int N = 1024;  // A cols, B rows
    const int K = 1024;  // B cols
    
    size_t size_A = M * N * sizeof(float);
    size_t size_B = N * K * sizeof(float);
    size_t size_C = M * K * sizeof(float);
    
    // Allocate host memory
    float *h_A = (float*)malloc(size_A);
    float *h_B = (float*)malloc(size_B);
    float *h_C_cpu = (float*)malloc(size_C);
    float *h_C_gpu = (float*)malloc(size_C);
    
    // Initialize matrices
    for (int i = 0; i < M * N; i++) h_A[i] = rand() / (float)RAND_MAX;
    for (int i = 0; i < N * K; i++) h_B[i] = rand() / (float)RAND_MAX;
    
    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);
    
    // Copy to device
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);
    
    // CPU Timing
    auto cpu_start = std::chrono::high_resolution_clock::now();
    matrixMulCPU(h_A, h_B, h_C_cpu, M, N, K);
    auto cpu_end = std::chrono::high_resolution_clock::now();
    double cpu_time = std::chrono::duration<double, std::milli>(cpu_end - cpu_start).count();
    
    // GPU Timing
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks((K + BLOCK_SIZE - 1) / BLOCK_SIZE, 
                   (M + BLOCK_SIZE - 1) / BLOCK_SIZE);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    matrixMulKernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, M, N, K);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float gpu_time;
    hipEventElapsedTime(&gpu_time, start, stop);
    
    // Copy result back
    hipMemcpy(h_C_gpu, d_C, size_C, hipMemcpyDeviceToHost);
    
    // Verify results
    float maxError = 0.0f;
    for (int i = 0; i < M * K; i++) {
        maxError = max(maxError, abs(h_C_cpu[i] - h_C_gpu[i]));
    }
    
    printf("Matrix Multiplication Results (%dx%d):\n", M, N);
    printf("CPU Time: %.2f milliseconds\n", cpu_time);
    printf("GPU Time: %.2f milliseconds\n", gpu_time);
    printf("Speedup: %.2fx\n", cpu_time/gpu_time);
    printf("Max Error: %e\n", maxError);
    
    // Cleanup
    free(h_A); free(h_B); free(h_C_cpu); free(h_C_gpu);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    hipEventDestroy(start); hipEventDestroy(stop);
    
    return 0;
}