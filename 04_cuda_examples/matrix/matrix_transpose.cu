#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>  // For CPU timing

// Function to verify the transposition result
bool verifyTransposition(const float* h_out, int rows, int cols) {
    bool isCorrect = true;
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            float expected = j * rows + i;
            float actual = h_out[i * cols + j];
            if (std::abs(expected - actual) > 1e-5) {
                std::cout << "Mismatch at (" << i << ", " << j << "): "
                          << "Expected " << expected << ", got " << actual << std::endl;
                isCorrect = false;
            }
        }
    }
    return isCorrect;
}

__global__ void transpose_conflict_free(const float* d_in, float* d_out, int rows, int cols) {
    __shared__ float tile[32][32+1];  // Add +1 to avoid bank conflicts

    int x = blockIdx.x * 32 + threadIdx.x;
    int y = blockIdx.y * 32 + threadIdx.y;

    int tid_x = threadIdx.x;
    int tid_y = threadIdx.y;

    if (x < cols && y < rows) {
        // Load data into shared memory
        tile[tid_y][tid_x] = d_in[y * cols + x];
    }
    __syncthreads();

    // Write transposed data from shared memory
    x = blockIdx.y * 32 + threadIdx.x;  // Swap block indices
    y = blockIdx.x * 32 + threadIdx.y;

    if (x < rows && y < cols) {
        d_out[y * rows + x] = tile[tid_x][tid_y];
    }
    __syncthreads();
}

// Naive CPU implementation of matrix transposition
void transpose_cpu(const float* h_in, float* h_out, int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            h_out[j * rows + i] = h_in[i * cols + j];
        }
    }
}

int main() {
    int rows = 4096;
    int cols = 4096;

    size_t size = rows * cols * sizeof(float);

    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    float* h_in = (float*)malloc(size);
    float* h_out = (float*)malloc(size);
    float* h_out_cpu = (float*)malloc(size);

    for (int i = 0; i < rows; ++i)
        for (int j = 0; j < cols; ++j)
            h_in[i * cols + j] = i * cols + j;

    float *d_in, *d_out;
    hipMalloc(&d_in, size);
    hipMalloc(&d_out, size);
    hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);

    dim3 block(32, 32);
    dim3 grid((cols + 31) / 32, (rows + 31) / 32);

    // GPU transposition
    hipEventRecord(startEvent, 0);
    transpose_conflict_free<<<grid, block>>>(d_in, d_out, rows, cols);
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);

    hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);

    // Calculate GPU elapsed time
    float gpuTime = 0;
    hipEventElapsedTime(&gpuTime, startEvent, stopEvent);

    // CPU transposition
    auto cpuStart = std::chrono::high_resolution_clock::now();
    transpose_cpu(h_in, h_out_cpu, rows, cols);
    auto cpuEnd = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> cpuTime = cpuEnd - cpuStart;

    // Verify the result
    bool transpositionCorrectGpu = verifyTransposition(h_out, rows, cols);
    bool transpositionCorrectCpu = verifyTransposition(h_out_cpu, rows, cols);

    if (transpositionCorrectGpu && transpositionCorrectCpu) {
        std::cout << "Matrix transposition verified successfully!" << std::endl;
    } else {
        std::cout << "Matrix transposition verification failed." << std::endl;
    }

    // Print execution times
    std::cout << "GPU execution time: " << gpuTime << " ms" << std::endl;
    std::cout << "CPU execution time: " << cpuTime.count() << " ms" << std::endl;

    // Calculate speedup
    double speedup = cpuTime.count() / gpuTime;
    std::cout << "Speedup (CPU vs GPU): " << speedup << "x" << std::endl;

    free(h_in);
    free(h_out);
    free(h_out_cpu);
    hipFree(d_in);
    hipFree(d_out);
    return 0;
}
