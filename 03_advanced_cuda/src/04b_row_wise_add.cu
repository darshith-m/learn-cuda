#include <iostream>
#include <hip/hip_runtime.h>

#define N 4096  // Size of the matrix

__global__ void matrixAddRowMajor(float *A, float *B, float *C, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < width && col < width) {
        int index = row * width + col;
        C[index] = A[index] + B[index];
    }
}

int main() {
    int size = N * N * sizeof(float);
    
    // Allocate memory for matrices A, B, C
    float *h_A = new float[N * N];
    float *h_B = new float[N * N];
    float *h_C = new float[N * N];
    
    // Initialize matrices
    for (int i = 0; i < N * N; ++i) {
        h_A[i] = 1.0f;  // Fill with 1
        h_B[i] = 2.0f;  // Fill with 2
    }

    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Set up CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Launch kernel with appropriate block size
    dim3 block(16, 16);
    dim3 grid((N + block.x - 1) / block.x, (N + block.y - 1) / block.y);
    
    hipEventRecord(start);
    matrixAddRowMajor<<<grid, block>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    float timeRowMajor;
    hipEventElapsedTime(&timeRowMajor, start, stop);  // Time in milliseconds
    std::cout << "Row-major matrix addition completed in " << timeRowMajor << " ms" << std::endl;

    // Cleanup
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
