#include <hip/hip_runtime.h>
#include <iostream>

__global__ void simpleKernel(float *data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] = data[idx] * data[idx]; // Example operation: square the value
    }
}

int main() {
    const int dataSize = 1 << 20; // 1 million elements
    const int blockSize = 256;
    const int chunkSize = dataSize / 4; // Breaking the data into 4 chunks
    const int numBlocks = (chunkSize + blockSize - 1) / blockSize;

    // Allocate host and device memory
    float *h_data = new float[dataSize];
    float *d_data1, *d_data2, *d_data3, *d_data4; // Device chunks
    hipStream_t stream1, stream2, stream3, stream4;

    // Allocate memory for device chunks
    hipMalloc((void **)&d_data1, chunkSize * sizeof(float));
    hipMalloc((void **)&d_data2, chunkSize * sizeof(float));
    hipMalloc((void **)&d_data3, chunkSize * sizeof(float));
    hipMalloc((void **)&d_data4, chunkSize * sizeof(float));

    // Create streams
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);
    hipStreamCreate(&stream4);

    // Initialize host data
    for (int i = 0; i < dataSize; i++) {
        h_data[i] = static_cast<float>(i);
    }

    // Create CUDA events for timing
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    // Record the start event
    hipEventRecord(startEvent, 0);

    // Process the first chunk using stream1
    hipMemcpyAsync(d_data1, h_data, chunkSize * sizeof(float), hipMemcpyHostToDevice, stream1);
    simpleKernel<<<numBlocks, blockSize, 0, stream1>>>(d_data1, chunkSize);
    hipMemcpyAsync(h_data, d_data1, chunkSize * sizeof(float), hipMemcpyDeviceToHost, stream1);

    // Process the second chunk using stream2
    hipMemcpyAsync(d_data2, h_data + chunkSize, chunkSize * sizeof(float), hipMemcpyHostToDevice, stream2);
    simpleKernel<<<numBlocks, blockSize, 0, stream2>>>(d_data2, chunkSize);
    hipMemcpyAsync(h_data + chunkSize, d_data2, chunkSize * sizeof(float), hipMemcpyDeviceToHost, stream2);

    // Process the third chunk using stream3
    hipMemcpyAsync(d_data3, h_data + 2 * chunkSize, chunkSize * sizeof(float), hipMemcpyHostToDevice, stream3);
    simpleKernel<<<numBlocks, blockSize, 0, stream3>>>(d_data3, chunkSize);
    hipMemcpyAsync(h_data + 2 * chunkSize, d_data3, chunkSize * sizeof(float), hipMemcpyDeviceToHost, stream3);

    // Process the fourth chunk using stream4
    hipMemcpyAsync(d_data4, h_data + 3 * chunkSize, chunkSize * sizeof(float), hipMemcpyHostToDevice, stream4);
    simpleKernel<<<numBlocks, blockSize, 0, stream4>>>(d_data4, chunkSize);
    hipMemcpyAsync(h_data + 3 * chunkSize, d_data4, chunkSize * sizeof(float), hipMemcpyDeviceToHost, stream4);

    // Synchronize all streams
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);
    hipStreamSynchronize(stream3);
    hipStreamSynchronize(stream4);

    // Record the stop event
    hipEventRecord(stopEvent, 0);

    // Wait for the event to complete
    hipEventSynchronize(stopEvent);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, startEvent, stopEvent);

    // Clean up
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
    hipFree(d_data1);
    hipFree(d_data2);
    hipFree(d_data3);
    hipFree(d_data4);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipStreamDestroy(stream3);
    hipStreamDestroy(stream4);
    delete[] h_data;

    // Print execution time
    std::cout << "Execution time with streams: " << milliseconds << " ms\n";

    return 0;
}
