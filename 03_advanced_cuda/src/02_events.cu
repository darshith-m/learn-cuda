#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    const int N = 1 << 20; // 1M elements
    const size_t size = N * sizeof(float);

    // Allocate host memory
    float *h_A = new float[N];
    float *h_B = new float[N];
    float *h_C = new float[N];

    // Initialize host arrays
    for (int i = 0; i < N; ++i) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i * 2);
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Launch the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Record the start event
    hipEventRecord(start);

    // Kernel execution
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Record the stop event
    hipEventRecord(stop);

    // Wait for the stop event to complete
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "Kernel execution time: " << milliseconds << " ms\n";

    // Copy results from device to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Clean up
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}
