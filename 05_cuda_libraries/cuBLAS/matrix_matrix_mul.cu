#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define M 3   // Rows of A and C
#define K 4   // Columns of A, rows of B
#define N 2   // Columns of B and C

void printMatrix(float *arr, int rows, int cols) {
    for(int i = 0; i < rows; i++) {
        for(int j = 0; j < cols; j++) {
            printf("%.2f ", arr[i * cols + j]);
        }
        printf("\n");
    }
}

int main() {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Host matrices
    float *h_A = (float*)malloc(M * K * sizeof(float));
    float *h_B = (float*)malloc(K * N * sizeof(float));
    float *h_C = (float*)malloc(M * N * sizeof(float));

    // Initialize matrices
    for(int i = 0; i < M * K; i++) h_A[i] = i + 1.0f;
    for(int i = 0; i < K * N; i++) h_B[i] = i + 1.0f;

    // Device matrices
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, M * K * sizeof(float));
    hipMalloc(&d_B, K * N * sizeof(float));
    hipMalloc(&d_C, M * N * sizeof(float));

    // Copy to device
    hipMemcpy(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice);

    // Perform matrix multiplication: C = alpha*A*B + beta*C
    float alpha = 1.0f;
    float beta = 0.0f;
    
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                M, N, K,
                &alpha,
                d_A, M,    // Leading dimension of A
                d_B, K,    // Leading dimension of B
                &beta,
                d_C, M);   // Leading dimension of C

    // Copy result back to host
    hipMemcpy(h_C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

    // Print results
    printf("Matrix A (%dx%d):\n", M, K);
    printMatrix(h_A, M, K);
    printf("\nMatrix B (%dx%d):\n", K, N);
    printMatrix(h_B, K, N);
    printf("\nResult C = A*B (%dx%d):\n", M, N);
    printMatrix(h_C, M, N);

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    hipblasDestroy(handle);

    return 0;
}