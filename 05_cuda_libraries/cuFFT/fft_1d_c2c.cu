#include <iostream>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

#define NX 4  // Number of rows
#define NY 4  // Number of columns

int main() {
    // Create 2D FFT plan for complex-to-complex transform
    hipfftHandle plan;
    hipfftPlan2d(
        &plan,      // Handle to FFT plan
        NX,         // Size of transform in X dimension
        NY,         // Size of transform in Y dimension
        HIPFFT_C2C   // Transform type: Complex to Complex
    );

    // Allocate device memory for input and output
    hipfftComplex* d_input;
    hipfftComplex* d_output;
    hipMalloc((void**)&d_input, sizeof(hipfftComplex) * NX * NY);   // Input array
    hipMalloc((void**)&d_output, sizeof(hipfftComplex) * NX * NY);  // Output array

    // Execute complex-to-complex FFT transform
    hipfftExecC2C(
        plan,           // FFT plan handle
        d_input,        // Input array (complex)
        d_output,       // Output array (complex)
        HIPFFT_FORWARD   // Transform direction (CUFFT_FORWARD or CUFFT_INVERSE)
    );

    // Note on memory layout:
    // 2D data is stored in row-major order
    // Index (i,j) maps to i*NY + j in linear memory
    // For C2C transforms, output size equals input size (NX * NY)

    // Clean up
    hipFree(d_input);
    hipFree(d_output);
    hipfftDestroy(plan);

    return 0;
}
