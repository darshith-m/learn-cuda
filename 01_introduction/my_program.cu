#include <iostream>
#include <hip/hip_runtime.h>

#define N 1000

__global__ void vectorAdd(const float* A, const float* B, float* C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;
    size_t size = N * sizeof(float);

    // Allocate host memory
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);

    // Initialize host arrays
    for (int i = 0; i < N; ++i) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i * 2);
    }

    // Allocate device memory
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result from device to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify result
    for (int i = 0; i < N; ++i) {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
            std::cerr << "Result verification failed at element " << i << std::endl;
            exit(EXIT_FAILURE);
        }
    }

    std::cout << "Test PASSED" << std::endl;

    // Free device and host memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
